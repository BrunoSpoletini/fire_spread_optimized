
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addKernel(int *a, int *b, int *result) {
    *result = *a + *b;
}

int main() {
    int h_a = 1, h_b = 2, h_result = 0;
    int *d_a, *d_b, *d_result;

    // Allocate device memory
    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_result, sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with 1 thread
    addKernel<<<1, 1>>>(d_a, d_b, d_result);
    
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    // Show result
    printf("%d + %d = %d\n", h_a, h_b, h_result);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
