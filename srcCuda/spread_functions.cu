#include "hip/hip_runtime.h"
#include "spread_functions.hpp"

#define _USE_MATH_DEFINES
#include <cmath>
#include <random>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "fires.hpp"
#include "landscape.hpp"

// Helper function for CUDA error checking
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)

__device__ float spread_probability(
    const Cell& burning, const Cell& neighbour, SimulationParams params, float angle,
    float distance, float elevation_mean, float elevation_sd, float upper_limit = 1.0
) {

  float slope_term = sin(atan((neighbour.elevation - burning.elevation) / distance));
  float wind_term = cos(angle - burning.wind_direction);
  float elev_term = (neighbour.elevation - elevation_mean) / elevation_sd;
  float linpred = params.independent_pred;

  if (neighbour.vegetation_type == SUBALPINE) {
    linpred += params.subalpine_pred;
  } else if (neighbour.vegetation_type == WET) {
    linpred += params.wet_pred;
  } else if (neighbour.vegetation_type == DRY) {
    linpred += params.dry_pred;
  }

  linpred += params.fwi_pred * neighbour.fwi;
  linpred += params.aspect_pred * neighbour.aspect;
  linpred += wind_term * params.wind_pred + elev_term * params.elevation_pred +
             slope_term * params.slope_pred;

             float prob = upper_limit / (1 + exp(-linpred));

  return prob;
}

// CUDA kernel for spread probability calculation
// Each thread processes one cell, if some of its neighbors are burning,
// it calculates the probability of burning itself based on the parameters
__global__ void calculate_spread_probabilities(
    const Cell* __restrict__ landscape,
    const unsigned short* __restrict__ burned_ids,
    unsigned int burned_size,
    unsigned int n_col,
    unsigned int n_row,
    const SimulationParams* params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float upper_limit,
    hiprandState* states,
    bool* burned_bin,
    char* cell_states_initial,
    char* cell_states_final
) {
    const float angles[8] = { M_PI * 3 / 4, M_PI, M_PI * 5 / 4, M_PI / 2, M_PI * 3 / 2,
                              M_PI / 4,     0,    M_PI * 7 / 4 };
    const int moves[8][2] = { { -1, -1 }, { -1, 0 }, { -1, 1 }, { 0, -1 },
                              { 0, 1 },   { 1, -1 }, { 1, 0 },  { 1, 1 } };

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop
    if (idx >= (n_col * n_row)) return; // Ensure we don't exceed the number of cells in the grid
    
    // Get the current cell coordinates
    unsigned int cell_x = idx % n_col;
    unsigned int cell_y = idx / n_col;
    const Cell& current_cell = landscape[cell_y * n_col + cell_x];


    // // Get burning cell coordinates
    // unsigned int burning_cell_0 = burning_cells[idx * 2];
    // unsigned int burning_cell_1 = burning_cells[idx * 2 + 1];
    // const Cell& burning_cell = landscape[burning_cell_1 * n_col + burning_cell_0];

    // // Get the random state for this cell
    // unsigned int cell_idx = burning_cell_1 * n_col + burning_cell_0;
    // hiprandState localState = states[cell_idx];

    // // Process each neighbor
    // for (int n = 0; n < 8; n++) {
    //     int neighbor_x = burning_cell_0 + moves[n][0];
    //     int neighbor_y = burning_cell_1 + moves[n][1];

    //     // Check if neighbor is in range
    //     if (neighbor_x < 0 || neighbor_x >= n_col || neighbor_y < 0 || neighbor_y >= n_row) {
    //         continue;
    //     }

    //     // Check if already burned
    //     if (burned_bin[neighbor_y * n_col + neighbor_x]) {
    //         continue;
    //     }

    //     const Cell& neighbor = landscape[neighbor_y * n_col + neighbor_x];
    //     if (!neighbor.burnable) {
    //         continue;
    //     }

    //     // Calculate spread probability using the helper function
    //     float prob = spread_probability(burning_cell, neighbor, *params, angles[n], 
    //                                   distance, elevation_mean, elevation_sd, upper_limit);

    //     // Random number generation and burn decision
    //     float random_value = hiprand_uniform(&localState);

    //     if (random_value < prob) {
    //         // Atomically add new burned cell
    //         unsigned int new_idx = atomicAdd(n_new_burned, 1);
    //         new_burned_cells[new_idx * 2] = neighbor_x;
    //         new_burned_cells[new_idx * 2 + 1] = neighbor_y;
    //     }
    // }

    // // Save the updated random state
    // states[cell_idx] = localState;
}

// Initialize random states
__global__ void init_random_states(hiprandState* states, unsigned int seed) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed + idx, 0, 0, &states[idx]);
}

Fire simulate_fire(
    const Landscape& landscape, const std::vector<std::pair<unsigned int, unsigned int>>& ignition_cells,
    SimulationParams params, float distance, float elevation_mean, float elevation_sd,
    float upper_limit = 1.0
) {
    // Use CUDA timing instead of OpenMP
    hipEvent_t start_event, stop_event;
    CUDA_CHECK(hipEventCreate(&start_event));
    CUDA_CHECK(hipEventCreate(&stop_event));
    CUDA_CHECK(hipEventRecord(start_event));

    unsigned int n_row = landscape.height;
    unsigned int n_col = landscape.width;

    // Initialize host arrays
    // std::vector<std::pair<unsigned int, unsigned int>> burned_ids;
    // std::vector<unsigned int> burned_ids_steps;
    //Matrix<bool> burned_bin(n_col, n_row);

    // Host variables
    Matrix<bool> burned_bin(n_col, n_row);
    char* cell_states_initial_h = new char[n_col * n_row];
    char* cell_states_final_h = new char[n_col * n_row];
    bool h_burned = true;
    unsigned int h_burned_size = 0;
    std::vector<std::pair<unsigned int, unsigned int>> h_burned_ids;
    std::vector<unsigned int> h_burned_ids_steps;
    size_t landscape_size = n_col * n_row * sizeof(Cell);

    // Device variables
    bool burned_d;
    char* cell_states_initial_d = nullptr;
    char* cell_states_final_d = nullptr;
    unsigned int d_burned_size = nullptr;
    unsigned short* d_burned_ids = nullptr;
    unsigned int* d_burned_ids_steps = nullptr;
    Cell* d_landscape = nullptr;
    hiprandState* d_states = nullptr;
    SimulationParams* d_params = nullptr;

    // Variables sizes
    size_t d_burned_ids_size = 2 * n_col * n_row * sizeof(unsigned short);
    size_t d_burned_ids_steps_size = (n_col * n_row) * sizeof(unsigned int);

    // Allocate device memory with error checking
    CUDA_CHECK(hipMalloc(&cell_states_initial_d, n_col * n_row * sizeof(char)));
    CUDA_CHECK(hipMalloc(&cell_states_final_d, n_col * n_row * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_burned_size, sizeof(int)));
    CUDA_CHECK(hipMalloc(&burned_d, sizeof(bool)));
    CUDA_CHECK(hipMalloc(&d_burned_ids, d_burned_ids_size));
    CUDA_CHECK(hipMalloc(&d_burned_ids_steps, d_burned_ids_steps_size));
    CUDA_CHECK(hipMalloc(&d_landscape, landscape_size));
    CUDA_CHECK(hipMalloc(&d_states, n_col * n_row * sizeof(hiprandState)));
    CUDA_CHECK(hipMalloc(&d_params, sizeof(SimulationParams)));

    // Initialize device memory to zero
    CUDA_CHECK(hipMemset(d_burned_ids, 0, d_burned_ids_size));
    CUDA_CHECK(hipMemset(d_burned_ids_steps, 0, d_burned_ids_steps_size));

    // Initialize cell states in HOST
    for (unsigned int i = 0; i < n_col; i++) {
        for (unsigned int j = 0; j < n_row; j++) {
            cell_states_initial_h[j * n_col + i] = 'U'; 
        }
    }

    // Add ignition cells
    for (const auto& cell : ignition_cells) {
        cell_states_initial_h[cell.second * n_col + cell.first] = 'B';
        h_burned_size++;
        h_burned_ids.push_back(cell);
    }
    h_burned_ids_steps.push_back(h_burned_size);

    // Copy burned_ids to device
    short int* burned_ids_temp = new short int[2 * h_burned_size];
    for (unsigned int i = 0; i < h_burned_size; i++) {
        burned_ids_temp[2*i] = h_burned_ids[i].first;
        burned_ids_temp[2*i+1] = h_burned_ids[i].second;
    }
    CUDA_CHECK(hipMemcpy(&d_burned_ids, burned_ids_temp, 2 * h_burned_size * sizeof(unsigned int), hipMemcpyHostToDevice));
    //CUDA_CHECK(hipMemcpy(&d_burned_ids_steps, burned_ids_steps.back(), sizeof(unsigned int), hipMemcpyHostToDevice));
    delete[] burned_ids_temp;

    // Copy initial cell states to DEVICE    
    CUDA_CHECK(hipMemcpy(cell_states_initial_d, cell_states_initial_h, n_col * n_row * sizeof(char), hipMemcpyHostToDevice));

    // Copy params to device
    CUDA_CHECK(hipMemcpy(d_params, &params, sizeof(SimulationParams), hipMemcpyHostToDevice));

    // Copy landscape to device
    Cell* landscape_data = new Cell[n_col * n_row];
    if (!landscape_data) {
        fprintf(stderr, "Failed to allocate landscape_data\n");
        exit(1);
    }
    for (unsigned int i = 0; i < n_col; i++) {
        for (unsigned int j = 0; j < n_row; j++) {
            landscape_data[j * n_col + i] = landscape[{i, j}];
        }
    }
    CUDA_CHECK(hipMemcpy(d_landscape, landscape_data, landscape_size, hipMemcpyHostToDevice));

    // Initialize random states
    int block_size = 256;
    int num_blocks = (n_col * n_row + block_size - 1) / block_size;
    init_random_states<<<num_blocks, block_size>>>(d_states, time(NULL));
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    unsigned int iteration = 1;

    //short int d_last_burned_size = d_burned_size;
    
    while (h_burned) {

        // Launch kernel
        calculate_spread_probabilities<<<num_blocks, block_size>>>(
            d_landscape, d_burned_ids, d_burned_size,
            n_col, n_row, d_params, distance, elevation_mean, elevation_sd,
            upper_limit, d_states, burned_d, cell_states_initial_d, cell_states_final_d
        );

        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());


        // Add the number of cells that burned in the last iteration to burned_ids_steps
        int old_burned_size = h_burned_size;
        CUDA_CHECK(hipMemcpy(&h_burned_size, d_burned_size, sizeof(unsigned int), hipMemcpyDeviceToHost));
        if (h_burned_size == old_burned_size) {
            // No new cells burned, exit loop
            h_burned = false;
            break;
        }
        h_burned_ids_steps.push_back(h_burned_size);
        

        // Swap initial and final states
        char* temp = cell_states_initial_d;
        cell_states_initial_d = cell_states_final_d;
        cell_states_final_d = temp;

        // Copy burned flag back to host
        CUDA_CHECK(hipMemcpy(&h_burned, burned_d, sizeof(bool), hipMemcpyDeviceToHost))

        // Agregar celdas incendiadas a steps
        CUDA_CHECK(hipMemcpy(d_burned_ids_steps + iteration * sizeof(unsigned short), &d_burned_iteration, sizeof(short int), hipMemcpyDeviceToDevice));

        // PASAR QUEMANDOSE A QUEMADOS CAPAZ

        // // --- DEBUG ---
        // // Get number of new burned cells
        // CUDA_CHECK(hipMemcpy(&n_new_burned, d_n_new_burned, sizeof(unsigned int), hipMemcpyDeviceToHost));
        // CUDA_CHECK(hipDeviceSynchronize());

        // // Debug print
        // fprintf(stderr, "n_new_burned = %u\n", n_new_burned);
        // if (n_new_burned > 2 * n_col * n_row) {
        //     fprintf(stderr, "ERROR: n_new_burned (%u) exceeds max possible cells (%u)!\n", n_new_burned, 2 * n_col * n_row);
        //     exit(1);
        // }
        // // --- END OF DEBUG ---

        ++iteration;
    }

    CUDA_CHECK(hipMemcpy(&cell_states_final_h, cell_states_final_d, n_col * n_row * sizeof(char), hipMemcpyDeviceToHost));
    for (unsigned int i = 0; i < n_col * n_row; i++) {
        unsigned int x = i % n_col;
        unsigned int y = i / n_col;
        burned_bin[x][y] = (cell_states_final_h[i] != 'U');
    }

    short int* h_burned_ids_aux = new short int[2 * n_col * n_row];
    CUDA_CHECK(hipMemcpy(&h_burned_ids_aux, d_burned_ids, d_burned_ids_size, hipMemcpyDeviceToHost));
    for (unsigned int i = 0; i < 2 * n_col * n_row; i += 2) {
        h_burned_ids.push_back({h_burned_ids_aux[i], h_burned_ids_aux[i + 1]});
    }

    short int* h_burned_ids_steps_aux = new short int[n_col * n_row];
    CUDA_CHECK(hipMemcpy(&h_burned_ids_steps_aux, d_burned_ids_steps, d_burned_ids_steps_size, hipMemcpyDeviceToHost));
    for (unsigned int i = 0; i < n_col * n_row; i++)
        h_burned_ids_steps.push_back(h_burned_ids_steps_aux[i]);

    // Free device memory
    if (d_landscape) CUDA_CHECK(hipFree(d_landscape));
    if (d_states) CUDA_CHECK(hipFree(d_states));
    if (d_params) CUDA_CHECK(hipFree(d_params));
    if (cell_states_initial_d) CUDA_CHECK(hipFree(cell_states_initial_d));
    if (cell_states_final_d) CUDA_CHECK(hipFree(cell_states_final_d));
    if (burned_d) CUDA_CHECK(hipFree(burned_d));
    if (d_burned_ids) CUDA_CHECK(hipFree(d_burned_ids));
    if (d_burned_ids_steps) CUDA_CHECK(hipFree(d_burned_ids_steps));
    if (d_burned_size) CUDA_CHECK(hipFree(d_burned_size));
    
    // Get elapsed time using CUDA events
    CUDA_CHECK(hipEventRecord(stop_event));
    CUDA_CHECK(hipEventSynchronize(stop_event));
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start_event, stop_event));
    double seconds = milliseconds / 1000.0;

    // fprintf(stderr, "Celdas incendiadas: %ld\n", burned_ids.size());
    // fprintf(stderr, "celdas incendiadas por microsegundo: %lf\n", burned_ids.size() / (1E06 * seconds));

    // Clean up CUDA events
    CUDA_CHECK(hipEventDestroy(start_event));
    CUDA_CHECK(hipEventDestroy(stop_event));

    delete[] landscape_data;
    delete[] h_burned_ids_aux;
    delete[] h_burned_ids_steps_aux;

    return { n_col, n_row, burned_bin, h_burned_ids, h_burned_ids_steps };
}
