#include "hip/hip_runtime.h"
#include "spread_functions.hpp"

#define _USE_MATH_DEFINES
#include <cmath>
#include <random>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "fires.hpp"
#include "landscape.hpp"

// Helper function for CUDA error checking
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)

__device__ float spread_probability(
    const Cell& burning, const Cell& neighbour, SimulationParams params, float angle,
    float distance, float elevation_mean, float elevation_sd, float upper_limit = 1.0
) {

  float slope_term = sin(atan((neighbour.elevation - burning.elevation) / distance));
  float wind_term = cos(angle - burning.wind_direction);
  float elev_term = (neighbour.elevation - elevation_mean) / elevation_sd;
  float linpred = params.independent_pred;

  if (neighbour.vegetation_type == SUBALPINE) {
    linpred += params.subalpine_pred;
  } else if (neighbour.vegetation_type == WET) {
    linpred += params.wet_pred;
  } else if (neighbour.vegetation_type == DRY) {
    linpred += params.dry_pred;
  }

  linpred += params.fwi_pred * neighbour.fwi;
  linpred += params.aspect_pred * neighbour.aspect;
  linpred += wind_term * params.wind_pred + elev_term * params.elevation_pred +
             slope_term * params.slope_pred;

             float prob = upper_limit / (1 + exp(-linpred));

  return prob;
}

// CUDA kernel for spread probability calculation
__global__ void calculate_spread_probabilities(
    const Cell* __restrict__ landscape,
    const bool* __restrict__ burned_bin,
    const unsigned int* __restrict__ burning_cells,
    unsigned int n_burning_cells,
    unsigned int n_col,
    unsigned int n_row,
    const SimulationParams* params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float upper_limit,
    hiprandState* states,
    unsigned int* new_burned_cells,
    unsigned int* n_new_burned
) {
    const float angles[8] = { M_PI * 3 / 4, M_PI, M_PI * 5 / 4, M_PI / 2, M_PI * 3 / 2,
                              M_PI / 4,     0,    M_PI * 7 / 4 };
    const int moves[8][2] = { { -1, -1 }, { -1, 0 }, { -1, 1 }, { 0, -1 },
                              { 0, 1 },   { 1, -1 }, { 1, 0 },  { 1, 1 } };

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_burning_cells) return;

    // Get burning cell coordinates
    unsigned int burning_cell_0 = burning_cells[idx * 2];
    unsigned int burning_cell_1 = burning_cells[idx * 2 + 1];
    const Cell& burning_cell = landscape[burning_cell_1 * n_col + burning_cell_0];

    // Get the random state for this cell
    unsigned int cell_idx = burning_cell_1 * n_col + burning_cell_0;
    hiprandState localState = states[cell_idx];

    // Process each neighbor
    for (int n = 0; n < 8; n++) {
        int neighbor_x = burning_cell_0 + moves[n][0];
        int neighbor_y = burning_cell_1 + moves[n][1];

        // Check if neighbor is in range
        if (neighbor_x < 0 || neighbor_x >= n_col || neighbor_y < 0 || neighbor_y >= n_row) {
            continue;
        }

        // Check if already burned
        if (burned_bin[neighbor_y * n_col + neighbor_x]) {
            continue;
        }

        const Cell& neighbor = landscape[neighbor_y * n_col + neighbor_x];
        if (!neighbor.burnable) {
            continue;
        }

        // Calculate spread probability using the helper function
        float prob = spread_probability(burning_cell, neighbor, *params, angles[n], 
                                      distance, elevation_mean, elevation_sd, upper_limit);

        // Random number generation and burn decision
        float random_value = hiprand_uniform(&localState);

        if (random_value < prob) {
            // Atomically add new burned cell
            unsigned int new_idx = atomicAdd(n_new_burned, 1);
            new_burned_cells[new_idx * 2] = neighbor_x;
            new_burned_cells[new_idx * 2 + 1] = neighbor_y;
        }
    }

    // Save the updated random state
    states[cell_idx] = localState;
}

// Initialize random states
__global__ void init_random_states(hiprandState* states, unsigned int seed) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed + idx, 0, 0, &states[idx]);
}

Fire simulate_fire(
    const Landscape& landscape, const std::vector<std::pair<unsigned int, unsigned int>>& ignition_cells,
    SimulationParams params, float distance, float elevation_mean, float elevation_sd,
    float upper_limit = 1.0
) {
    // Use CUDA timing instead of OpenMP
    hipEvent_t start_event, stop_event;
    CUDA_CHECK(hipEventCreate(&start_event));
    CUDA_CHECK(hipEventCreate(&stop_event));
    CUDA_CHECK(hipEventRecord(start_event));

    unsigned int n_row = landscape.height;
    unsigned int n_col = landscape.width;

    // Initialize host arrays
    std::vector<std::pair<unsigned int, unsigned int>> burned_ids;
    std::vector<unsigned int> burned_ids_steps;
    Matrix<bool> burned_bin(n_col, n_row);

    // Add ignition cells
    for (const auto& cell : ignition_cells) {
        burned_ids.push_back(cell);
        burned_bin[{cell.first, cell.second}] = true;
    }
    burned_ids_steps.push_back(ignition_cells.size());

    // Allocate device memory with error checking
    Cell* d_landscape = nullptr;
    bool* d_burned_bin = nullptr;
    unsigned int* d_burning_cells = nullptr;
    unsigned int* d_new_burned_cells = nullptr;
    unsigned int* d_n_new_burned = nullptr;
    hiprandState* d_states = nullptr;
    SimulationParams* d_params = nullptr;

    size_t landscape_size = n_col * n_row * sizeof(Cell);
    size_t burned_bin_size = n_col * n_row * sizeof(bool);
    size_t burning_cells_size = 2 * n_col * n_row * sizeof(unsigned int);
    size_t new_burned_cells_size = 2 * n_col * n_row * sizeof(unsigned int);

    CUDA_CHECK(hipMalloc(&d_landscape, landscape_size));
    CUDA_CHECK(hipMalloc(&d_burned_bin, burned_bin_size));
    CUDA_CHECK(hipMalloc(&d_burning_cells, burning_cells_size));
    CUDA_CHECK(hipMalloc(&d_new_burned_cells, new_burned_cells_size));
    CUDA_CHECK(hipMalloc(&d_n_new_burned, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_states, n_col * n_row * sizeof(hiprandState)));
    CUDA_CHECK(hipMalloc(&d_params, sizeof(SimulationParams)));

    // Initialize device memory to zero
    CUDA_CHECK(hipMemset(d_burned_bin, 0, burned_bin_size));
    CUDA_CHECK(hipMemset(d_burning_cells, 0, burning_cells_size));
    CUDA_CHECK(hipMemset(d_new_burned_cells, 0, new_burned_cells_size));
    CUDA_CHECK(hipMemset(d_n_new_burned, 0, sizeof(unsigned int)));

    // Copy params to device
    CUDA_CHECK(hipMemcpy(d_params, &params, sizeof(SimulationParams), hipMemcpyHostToDevice));

    // Copy landscape to device
    Cell* landscape_data = new Cell[n_col * n_row];
    if (!landscape_data) {
        fprintf(stderr, "Failed to allocate landscape_data\n");
        exit(1);
    }

    for (unsigned int i = 0; i < n_col; i++) {
        for (unsigned int j = 0; j < n_row; j++) {
            landscape_data[j * n_col + i] = landscape[{i, j}];
        }
    }
    CUDA_CHECK(hipMemcpy(d_landscape, landscape_data, landscape_size, hipMemcpyHostToDevice));
    delete[] landscape_data;

    // Copy burned_bin to device
    bool* burned_bin_data = new bool[n_col * n_row];
    if (!burned_bin_data) {
        fprintf(stderr, "Failed to allocate burned_bin_data\n");
        exit(1);
    }

    for (unsigned int i = 0; i < n_col; i++) {
        for (unsigned int j = 0; j < n_row; j++) {
            burned_bin_data[j * n_col + i] = burned_bin[{i, j}];
        }
    }
    CUDA_CHECK(hipMemcpy(d_burned_bin, burned_bin_data, burned_bin_size, hipMemcpyHostToDevice));
    delete[] burned_bin_data;

    // Initialize random states
    int block_size = 256;
    int num_blocks = (n_col * n_row + block_size - 1) / block_size;
    init_random_states<<<num_blocks, block_size>>>(d_states, time(NULL));
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    unsigned int current_start = 0;
    unsigned int current_end = ignition_cells.size();
    unsigned int burning_size = current_end;

    while (burning_size > 0) {
        // Verify indices are valid
        if (current_start > burned_ids.size()) {
            fprintf(stderr, "Error: current_start (%u) exceeds burned_ids size (%zu)\n", 
                    current_start, burned_ids.size());
            break;
        }

        // Calculate actual burning size based on available cells
        burning_size = std::min(burning_size, static_cast<unsigned int>(burned_ids.size() - current_start));
        if (burning_size == 0) break;

        // Copy current burning cells to device
        unsigned int* current_burning = new unsigned int[2 * burning_size];
        if (!current_burning) {
            fprintf(stderr, "Failed to allocate current_burning\n");
            exit(1);
        }

        // Copy burning cells with bounds checking
        for (unsigned int i = 0; i < burning_size; i++) {
            unsigned int idx = current_start + i;
            if (idx >= burned_ids.size()) {
                fprintf(stderr, "Error: Index %u out of bounds in burned_ids (size: %zu)\n", 
                        idx, burned_ids.size());
                delete[] current_burning;
                exit(1);
            }
            current_burning[i * 2] = burned_ids[idx].first;
            current_burning[i * 2 + 1] = burned_ids[idx].second;
        }

        CUDA_CHECK(hipMemcpy(d_burning_cells, current_burning, 2 * burning_size * sizeof(unsigned int), hipMemcpyHostToDevice));
        delete[] current_burning;

        // Reset counter for new burned cells
        unsigned int zero = 0;
        CUDA_CHECK(hipMemcpy(d_n_new_burned, &zero, sizeof(unsigned int), hipMemcpyHostToDevice));

        // Launch kernel
        num_blocks = (burning_size + block_size - 1) / block_size;
        calculate_spread_probabilities<<<num_blocks, block_size>>>(
            d_landscape, d_burned_bin, d_burning_cells, burning_size,
            n_col, n_row, d_params, distance, elevation_mean, elevation_sd,
            upper_limit, d_states, d_new_burned_cells, d_n_new_burned
        );
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Get number of new burned cells
        unsigned int n_new_burned = 0;
        CUDA_CHECK(hipMemcpy(&n_new_burned, d_n_new_burned, sizeof(unsigned int), hipMemcpyDeviceToHost));

        // Debug print
        //fprintf(stderr, "n_new_burned = %u\n", n_new_burned);
        if (n_new_burned > 2 * n_col * n_row) {
            fprintf(stderr, "ERROR: n_new_burned (%u) exceeds max possible cells (%u)!\n", n_new_burned, 2 * n_col * n_row);
            exit(1);
        }

        if (n_new_burned > 0) {
            // Copy new burned cells back to host
            unsigned int* new_burned = new unsigned int[2 * n_new_burned];
            if (!new_burned) {
                fprintf(stderr, "Failed to allocate new_burned\n");
                exit(1);
            }

            CUDA_CHECK(hipMemcpy(new_burned, d_new_burned_cells, 2 * n_new_burned * sizeof(unsigned int), hipMemcpyDeviceToHost));

            // Update burned_bin and burned_ids
            for (unsigned int i = 0; i < n_new_burned; i++) {
                unsigned int x = new_burned[i * 2];
                unsigned int y = new_burned[i * 2 + 1];
                
                // Strict bounds checking
                if (x >= n_col || y >= n_row) {
                    fprintf(stderr, "Warning: Out of bounds cell coordinates (%u, %u) with grid size (%u, %u)\n", 
                            x, y, n_col, n_row);
                    continue;
                }

                // Check if cell is already burned
                if (burned_bin[{x, y}]) {
                    continue;
                }

                burned_bin[{x, y}] = true;
                burned_ids.push_back({x, y});
            }
            delete[] new_burned;
        }

        // Update indices
        current_start = current_end;
        current_end = burned_ids.size();  // Use actual size instead of adding n_new_burned
        burning_size = n_new_burned;
        burned_ids_steps.push_back(current_end);
    }

    // Free device memory
    if (d_landscape) CUDA_CHECK(hipFree(d_landscape));
    if (d_burned_bin) CUDA_CHECK(hipFree(d_burned_bin));
    if (d_burning_cells) CUDA_CHECK(hipFree(d_burning_cells));
    if (d_new_burned_cells) CUDA_CHECK(hipFree(d_new_burned_cells));
    if (d_n_new_burned) CUDA_CHECK(hipFree(d_n_new_burned));
    if (d_states) CUDA_CHECK(hipFree(d_states));
    if (d_params) CUDA_CHECK(hipFree(d_params));

    // Get elapsed time using CUDA events
    CUDA_CHECK(hipEventRecord(stop_event));
    CUDA_CHECK(hipEventSynchronize(stop_event));
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start_event, stop_event));
    double seconds = milliseconds / 1000.0;

    fprintf(stderr, "Celdas incendiadas: %ld\n", burned_ids.size());
    fprintf(stderr, "celdas incendiadas por microsegundo: %lf\n", burned_ids.size() / (1E06 * seconds));

    // Clean up CUDA events
    CUDA_CHECK(hipEventDestroy(start_event));
    CUDA_CHECK(hipEventDestroy(stop_event));

    return { n_col, n_row, burned_bin, burned_ids, burned_ids_steps };
}
